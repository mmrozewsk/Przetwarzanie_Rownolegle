#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdio.h>
#include <time.h>
#include "math.h"

__global__ void generate_numbers(int range, int* numbers, int range_sqrt) {
    
    int beg = threadIdx.x * range / blockDim.x;
    int end = (threadIdx.x + 1) * range / blockDim.x - 1;

    for (int i = beg; i <= end; i++) {
        if (i > 1)
            numbers[i] = 1;
    }

    __syncthreads();
    if (threadIdx.x == blockDim.x - 1) {
        
        for (int i = 2; i <= range_sqrt; i++) {
            if (numbers[i] == 1) {
                int temp_val = i * 2;
                while (temp_val <= range) {
                    numbers[temp_val] = 0;
                    temp_val += i;
                }
            }
        }
    }

    int counter = 0;
    __syncthreads();
    for (int i = beg; i <= end; i++) {
        if (numbers[i] == 1) {
            counter += 1;
        }
    }

    __shared__ int counters[8];
    counters[threadIdx.x] = counter;
    __syncthreads();
    if (threadIdx.x == blockDim.x - 1) {
        counter = 0;
        for (int i = 0; i < blockDim.x; i++) {
            counter += counters[i];
        }
        printf("Primes in total: %d\n", counter);
    }

    __syncthreads();

}

int main() {
    int range = 100000000;
    int threads = 16;
    int range_sqrt = sqrt(range);
    clock_t start_time, end_time;
    
    double total_time;

    int* numbers;
    size_t mem = range * sizeof(int);
    hipMallocManaged(&numbers, mem);

    dim3 THREADS(threads, threads);
    start_time = clock();
    generate_numbers << <1, threads, threads * sizeof(int) >> > (range, numbers, range_sqrt);
    hipDeviceSynchronize();
    end_time = clock();
    total_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    printf("Time: %f, Range: %d, Threads:%d\n", total_time, range, threads);

    return 0;
}
