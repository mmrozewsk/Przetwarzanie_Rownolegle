#include "hip/hip_runtime.h"

#include "math.h"
#include <stdio.h>
#include <time.h>

int generate_numbers(int range, int range_sqrt) {
    int* numbers;
    size_t bytes = range * sizeof(int);
    hipMallocManaged(&numbers, bytes);

    for (int i = 2; i <= range; i++) {
        numbers[i] = 1;
    }
    
    for (int i = 2; i <= range_sqrt; i++) {
        if (numbers[i] == 1) {
            int temp_val = i * 2;
            while (temp_val <= range) {
                numbers[temp_val] = 0;
                temp_val += i;
            }
        }
    }

    int counter = 0;
    for (int i = 2; i <= range; i++) {
        if (numbers[i] == 1) {
            counter += 1;
        }
    }
    return counter;
}

int main(int argc, char* argv[]) {
    int range = atoi(argv[1]);
    int range_sqrt = sqrt(range);
    clock_t start, end;
    double total_time;

    //sito
    start = clock();
    int counter = 0;
    counter = generate_numbers(range, range_sqrt);
    end = clock();
    total_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Primes in total: %d\n", counter);
    printf("Time: %f, Range: %d\n", total_time, range);

    return 0;
}