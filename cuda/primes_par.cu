#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/device_functions.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include "math.h"

__global__ void generate_numbers(int range, int* numbers, int range_sqrt) {

    int beg = threadIdx.x * range / blockDim.x;
    int end = (threadIdx.x + 1) * range / blockDim.x - 1;

    //jedynki
    numbers[2] = 1;
    for (int i = beg; i <= end; i++) {
        if (i > 1 && i%2 == 1)
            numbers[i] = 1;
    }

    __syncthreads();
    //sito
    if (threadIdx.x == blockDim.x - 1) {

        for (int i = 3; i <= range_sqrt; i += 2) {
            if (numbers[i] == 1) {
                int temp_val = i * 2;
                while (temp_val <= range) {
                    numbers[temp_val] = 0;
                    temp_val += i;
                }
            }
        }
    }

    int counter = 0;
    __syncthreads();
    //zliczanie
    for (int i = beg; i <= end; i++) {
        if (numbers[i] == 1) {
            counter += 1;
        }
    }

    __shared__ int counters[8];
    counters[threadIdx.x] = counter;
    __syncthreads();
    //wynik
    if (threadIdx.x == blockDim.x - 1) {
        counter = 0;
        for (int i = 0; i < blockDim.x; i++) {
            counter += counters[i];
        }
        printf("Primes in total: %d\n", counter);
    }

    __syncthreads();

}

int main(int argc, char* argv[]) {
    int range = atoi(argv[1]);
    int threads = atoi(argv[2]);
    int range_sqrt = sqrt(range);
    clock_t start_time, end_time;

    double total_time;

    int* numbers;
    size_t mem = range * sizeof(int);
    hipMallocManaged(&numbers, mem);

    dim3 THREADS(threads, threads);
    start_time = clock();
    generate_numbers << <1, threads, threads * sizeof(int) >> > (range, numbers, range_sqrt);
    hipDeviceSynchronize();
    end_time = clock();
    total_time = (double)(end_time - start_time) / CLOCKS_PER_SEC;
    printf("Time: %f, Range: %d, Threads:%d\n", total_time, range, threads);

    return 0;
}